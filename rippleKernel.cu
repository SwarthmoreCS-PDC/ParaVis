#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "rippleKernel.h"

__global__ void kernel(color3 *ptr, int w, int h, int ticks);

void RippleKernel::update(ImageBuffer* img) {
  int tdim = 8;
  int w = img->width;
  int h = img->height;
  dim3 blocks((w+(tdim-1)) / tdim, (h+(tdim-1)) / tdim);
  dim3 threads_block(tdim, tdim);
  kernel<<<blocks, threads_block>>>(img->buffer, w, h, m_ticks);
  m_ticks += 2;
}

__global__ void kernel(color3 *ptr, int w, int h, int ticks) {
// map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * w;

  // compute distance from center of image
  float fx = x - w / 2;
  float fy = y - h / 2;
  float d = sqrtf(fx * fx + fy * fy);
  unsigned char grey = (unsigned char)(128.0f +
                                       127.0f * cos(d / 10.0f - ticks / 7.0f) /
                                           (d / 10.0f + 1.0f));
  ptr[offset].r = grey;
  ptr[offset].g = grey;
  ptr[offset].b = grey;
}
