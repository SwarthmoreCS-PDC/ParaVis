#include "cudaWrapper.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

CUDAWrapper::CUDAWrapper() : m_pbo_CUDA(nullptr){};

void CUDAWrapper::init() {
  /* deprecated as of CUDA 5.0 */
  // cudaGLSetGLDevice(0);
}

void CUDAWrapper::connect(GLuint buffID) {
  if (m_pbo_CUDA) {
    disconnect();
  }
  hipGraphicsGLRegisterBuffer(&m_pbo_CUDA, buffID,
                               hipGraphicsRegisterFlagsNone);
}

void CUDAWrapper::disconnect() {
  if (m_pbo_CUDA) {
    hipGraphicsUnregisterResource(m_pbo_CUDA);
    m_pbo_CUDA = nullptr;
  }
}

bool CUDAWrapper::copyToGPU(color3 *cpuBuff, int width, int height) {
  color3 *dev_img = this->map();
  hipError_t err;
  err = hipMemcpy(dev_img, cpuBuff, sizeof(color3) * width * height,
                   hipMemcpyHostToDevice);
  return (err == hipSuccess);
}

color3 *CUDAWrapper::map() {

  color3 *dev_pixBuffer;
  size_t numBytes;

  // Map buffer object for writing from CUDA
  hipGraphicsMapResources(1, &m_pbo_CUDA);
  hipGraphicsResourceGetMappedPointer((void **)&dev_pixBuffer, &numBytes,
                                       m_pbo_CUDA);
  return dev_pixBuffer;
}

void CUDAWrapper::unmap() {
  hipDeviceSynchronize(); // Make sure kernel is done
  // Return PBO to OpenGL control.
  hipGraphicsUnmapResources(1, &m_pbo_CUDA);
}
