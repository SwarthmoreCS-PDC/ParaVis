#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "juliaKernel.h"

__device__ int julia(int x, int y, int w, int h, float re, float im);
__global__ void kernel(uchar3 *ptr, int w, int h, float re, float im);


void JuliaKernel::run(void* buff, float re, float im) {

  dim3 grid(m_width, m_height);
  im += 0.2 * sin(m_ticks/20.);
  re += 0.3 * cos(m_ticks/17.);
  kernel<<<grid, 1>>>((uchar3*)buff, m_width, m_height, re, im);
  m_ticks = (m_ticks+1)%1234;
}

struct hipComplex {
  float r;
  float i;
  __device__ hipComplex(float a, float b) : r(a), i(b) {}
  __device__ float magnitude2(void) { return r * r + i * i; }
  __device__ hipComplex operator*(const hipComplex &a) {
    return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
  }
  __device__ hipComplex operator+(const hipComplex &a) {
    return hipComplex(r + a.r, i + a.i);
  }
};

__device__ int julia(int x, int y, int w, int h, float re, float im) {
  const float scale = 1.5;
  float jx = scale * (float)(w / 2 - x) / (h / 2);
  float jy = scale * (float)(h / 2 - y) / (h / 2);

  hipComplex c(re, im);
  hipComplex a(jx, jy);

  int i = 0;
  for (i = 0; i < 200; i++) {
    a = a * a + c;
    if (a.magnitude2() > 1000) {
      return 0;
    }
  }

  return 1;
}

__global__ void kernel(uchar3 *ptr, int w, int h, float re, float im) {
  // map from threadIdx/BlockIdx to pixel position
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x + y * gridDim.x;

  // now calculate the value at that position
  int juliaValue = julia(x, y, w, h, re, im);
  ptr[offset].x = 255 * juliaValue;
  ptr[offset].y = 0;
  ptr[offset].z = 64 * (1 - juliaValue);
}
